# include <stdio.h>
# include <stdlib.h>
# include <string.h>
//# include <hip/hip_runtime.h>
# include <time.h>

# include "sudoku.h"

void ini_location(loc *l, int i, int j, int val){
  l->i = i;
  l->j = j;
  l->val = val;
  l->on = val==0?0:1;
}

sudoku_puzzle* ini_puzzle(int id, FILE *f){
  sudoku_puzzle *s = (sudoku_puzzle *)malloc(sizeof(sudoku_puzzle));
  s->id = id;

  s->map2chr = (int **)malloc(sizeof(int*)*9);
  s->colpre = (int **)malloc(sizeof(int*)*9);
  s->rowpre = (int **)malloc(sizeof(int*)*9);
  s->solution = (int **)malloc(sizeof(int*)*9);
  s->blocks = (loc **)malloc(sizeof(loc*)*9);

  int i;
  for(i=0; i<9; i++){
    s->map2chr[i] = (int *)malloc(sizeof(int)*9);
    s->solution[i] = (int *)malloc(sizeof(int)*9);
    s->colpre[i] = (int *)malloc(sizeof(int)*9);
    s->rowpre[i] = (int *)malloc(sizeof(int)*9);
    s->blocks[i] = (loc *)malloc(sizeof(loc)*9);

    int j = 0;
    for (;j<9;j++){ //initial
      s->colpre[i][j] = 0;
      s->rowpre[i][j] = 0;
      s->map2chr[i][j] = -1;
    }
  }

  DEBUG("start scan...");

  int j = 0;
  char buf;
  int tmp;
  while (j<9){
    i = 0;
    while (i<9){
      if (fscanf(f, "%c", &buf)){
        if (buf == '\n')
          continue;
        tmp = atoi(&buf);
        s->solution[j][i] = tmp;
        if (tmp != 0){
          s->colpre[i][tmp-1] = 1;
          s->rowpre[j][tmp-1] = 1;
        }
        //printf("puzzle[%d][%d], blocks[%d][%d] ", j, i, j/3*3+i/3, j%3*3+i%3);
        ini_location(&(s->blocks[j/3*3+i/3][j%3*3+i%3]), j, i, tmp);
        //printf("%d\n",s->blocks[j/3*3+i/3][j%3*3+i%3].on);
        i++;
      }
   }
    j++;
  }
  DEBUG("finish scan!");

  return s;
}


void export_sudoku(char * r, sudoku_puzzle *s, FILE *sf){
  
  DEBUG("start print...");
  int i, j;
  for(i=0;i<9;i++){
    for(j=0;j<9;j++){
      int x = s->map2chr[i][j];
      if (x!=-1){ s->solution[i][j] = r[x]-'0'; }
      fprintf(sf, "%d", s->solution[i][j]);
    }
  fprintf(sf, "\n");
  }
  DEBUG("finish export!");
}

void clean_sudoku(sudoku_puzzle *s){
  int i = 0;
  for(;i<9;i++){
    free(s->map2chr[i]);
    free(s->blk2chr[i]);
    free(s->colpre[i]);
    free(s->rowpre[i]);
    free(s->solution[i]);
    free(s->blocks[i]);
    free(s->map[i]);
  }
  free(s->map2chr);
  free(s->colpre);
  free(s->rowpre);
  free(s->solution);
  free(s->blocks);
  free(s->map);
  free(s->blk2chr);
  free(s);
}
