#include "hip/hip_runtime.h"
# include <stdio.h>
# include <stdlib.h>
# include <time.h>
# include <hip/hip_runtime.h>
# include <hiprand/hiprand_kernel.h>

# include "sudoku.h"

# define N 128//population size
# define BLKN 1 //grid size
# define M_RATE 0.5 //mutation rate
# define MUTATION 3 //schocastic mutation set size
# define TOURNAMENT 3 //schocastic tourament set size-1
# define MAX_CHR 74 //maximum chromosome size, 81 -(3^2-1) for unique solution
# define FULL 160 //maximum score 162 = 9*9*2

/*I can't get the program finished if this is 162 exactly. 
 *Reasoning that there can't be 161 score solution, 160 gives the right answer. 
 *Please help if you know why it doesn't terminate if this is 162
*/

# define CUDA_ERROR_CHECK(error) {\
           e = error; \
           if (e != hipSuccess){ \
             DEBUG(hipGetErrorString(e)); \
             free_exit(f, sf, pool, s); \
             return 0;\
           }	\
         }


// constant mem
static __constant__ int d_map2chr[9][9]; //position on pannel to chromosome index
static __constant__ int d_colpre[9][9]; //column present bits
static __constant__ int d_rowpre[9][9]; //row present bits, val-1 --> 1 or 0
static __constant__ int d_blk2chr[9][2]; //blk2chr[i] is [i_start, i_length] of i th block

void free_exit(FILE *f, FILE *sf, char * pool, sudoku_puzzle *s);

void shuffle(int * a, int n);

char * scan_puzzle(sudoku_puzzle *s, int pool_size);

__device__ int sum(int * a, int n){
  int i = 0;
  int r = 0;
  for (;i<n;i++){
    r += a[i];
  }
  return r;
}

__device__ void mutation(hiprandState *s, char * chrom){

  int i, x, y;
  float m;

  for (i=0;i<9;i++){//iterate on blocks
      m = hiprand_uniform(s); //decide whether to mutate
      if (m < M_RATE){ continue; }

      else{// chose 2 from the block_th block

        m = hiprand_uniform(s);
        x = (int)(m*d_blk2chr[i][1])%d_blk2chr[i][1]; // offset in block  ->1
        m = hiprand_uniform(s);
        y = (int)(m*d_blk2chr[i][1])%d_blk2chr[i][1]; // offset in block  ->2

        //swap
        char tmp = chrom[d_blk2chr[i][0]+x];
        chrom[d_blk2chr[i][0]+x] = chrom[d_blk2chr[i][0]+y];
        chrom[d_blk2chr[i][0]+y] = tmp;
      }
  }

  return;
}

__device__ void evaluate(int * allcs, int * allrs, char * chrom, int off){

  int cs = 0; //sum for 3 column: off*3-off*3+3
  int rs = 0; //sum for 3 row: off*3-off*3+3
  int i,j;

  for (i=0;i<3;i++){

    int x = off*3+i; //x_th
    int mc[9], mr[9];

    // cp from constant to local: present bits for x_th ROW and COL
    for (j=0;j<9;j++){
      mc[j] = d_colpre[x][j]; 
      mr[j] = d_rowpre[x][j];
    }

    for (j=0;j<9;j++){
      if (d_map2chr[x][j]!=-1){ // this element(x_th, j) on chromosome?
        mr[chrom[d_map2chr[x][j]]-'0'-1] = 1; } //yes -> change present bit ROW x has it now!
      if (d_map2chr[j][x]!=-1){ // this element(j, x_th) on chromosome? 
        mc[chrom[d_map2chr[j][x]]-'0'-1] = 1; } //yes -> change present bit COL x has it now!
    }

    cs += sum(mc, 9);
    rs += sum(mr, 9);
  }
  allcs[off] = cs;
  allrs[off] = rs;
}

__device__ void cpy2global(char * to, char * in, int n){
  int i;
  for(i=0;i<n;i++){
    to[i] = in[i];
  }
  return;
}

__global__ void ini_device(hiprandState * state, unsigned long seed){
  int id = threadIdx.x + blockIdx.x*blockDim.x;
  hiprand_init(seed, id, 0, &state[id]);
}

__global__ void solve(hiprandState * globalState, char * pp, char * final, int chr_size){

  __shared__ char chrs[N][MAX_CHR]; //population pool, >=rank^2 -1 given --> 1 solution 
  __shared__ char cross[N][MAX_CHR]; //child working area during crossing

  __shared__ int cscores[N][3]; // column scores
  __shared__ int rscores[N][3]; // row scores
  __shared__ int total[N]; // total scores

  __shared__ int pa[N]; // parents' id

  __shared__ int going; // flag


  // initialize, prepare, load to share mem
  int i,j;
  int chr = threadIdx.x;
 
  going = -1;

  for (i=0;i<chr_size;i++){ 
      chrs[chr][i] =  pp[chr*chr_size+i]; 
  }

  float rdf;
  int rdi;
  hiprandState localState = globalState[threadIdx.x + blockIdx.x*blockDim.x];
  
  __syncthreads();

  while (going < 0){

    // calculate scores
    for(i=0;i<3;i++){ evaluate(cscores[chr], rscores[chr], chrs[chr], i); }
    total[chr] = sum(rscores[chr], 3) + sum(cscores[chr], 3);

    if (total[chr] >= FULL){
      going = chr;
      cpy2global(final, chrs[chr], chr_size);
      return;
    }
   
    // tournament selection
    pa[chr] = chr;

    for (i=1;i<TOURNAMENT;i++){
      rdf = hiprand_uniform( &localState );
      rdi = (int)(rdf*N)%N;

      pa[chr] = total[rdi] > total[pa[chr]] ? rdi:pa[chr]; // thread id for chr_th parent
    }
    
    __syncthreads();

    // cross
    int win; //col/row winner id
    int p1 = pa[chr];
    int p2 = pa[N-1-chr];

    if (chr > N/2){ // perform ROW CROSS
      for (i=0;i<3;i++){
        win = rscores[p1][i] > rscores[p2][i] ? p1:p2;
        int k;
        for (k=0; k<3; k++){
          for (j=0; j<d_blk2chr[i*3+k][1];j++){ // off*3+{0,1,2} blocks in a row
            cross[chr][d_blk2chr[i*3+k][0]+j] = chrs[win][d_blk2chr[i*3+k][0]+j];
        }} // cpy from shared: generate child
      }
    }
    else { // perform COL CROSS
      for (i=0;i<3;i++){ 
        win = cscores[p1][i] > cscores[p2][i] ? p1:p2;
        int k;
        for (k=0; k<3; k++){
          for (j=0; j<d_blk2chr[i+k*3][1]; j++){ // off+{0,3,6} blocks in a column
            	//blk2chr[block]  = {start, length}
            cross[chr][d_blk2chr[i+k*3][0]+j] = chrs[win][d_blk2chr[i+k*3][0]+j]; 
        }}
      }
    }
    // evaluate
    for(i=0;i<3;i++){ evaluate(cscores[chr], rscores[chr], cross[chr], i); } // evaluate on working copy
    total[chr] = sum(rscores[chr], 3) + sum(cscores[chr], 3);
    if (total[chr] >= FULL){
      going = chr;
      cpy2global(final, cross[chr], chr_size);
      return;
    }
      // mutation
    for(i=0;i<MUTATION;i++){
      int max_score = total[chr];
      int tmp;
      mutation(&localState, cross[chr]); //mutate
      for(i=0;i<3;i++){ evaluate(cscores[chr], rscores[chr], cross[chr], i); } // evaluate on working copy
      tmp = sum(rscores[chr], 3) + sum(cscores[chr], 3);
      if (tmp>=FULL){
        going = chr;
        cpy2global(final, cross[chr], chr_size);
      }
      if (tmp > max_score){
        max_score = tmp;
        for (j=0;j<chr_size;j++){ chrs[chr][j] = cross[chr][j]; } // copy to pool
      }
    }
  }

  //globalState[threadIdx.x + blockIdx.x*blockDim.x] = localState;

}

//-------------------------------------

int main(int argc, char *argv[]){

  FILE *f, *sf;
  f = fopen(argv[1], "r");

  if (f==NULL){
    DEBUG("usage: sudokusolver filename.in");
    exit(0);
  }

  int fname_size = strlen(argv[1]);
  char fname_out[fname_size+1];
  char *temp = ".sol";
  memcpy(fname_out, argv[1], fname_size-3);
  memcpy(fname_out+fname_size-3, temp, 4);

  sf = fopen(fname_out, "w");
  if (sf==NULL){
    DEBUG("unable to create output file\n");
    exit(0);
  }

  sudoku_puzzle *s = ini_puzzle(0, f);
  char * pool = scan_puzzle(s, N);

  hipError_t e;
  // constant
    // 1. generate continous 2D block local
  int blk2chr[9][2];
  int map2chr[9][9];
  int colpre[9][9];
  int rowpre[9][9];
  int i;

  for (i=0;i<9;i++){
    int j;
    blk2chr[i][0] = s->blk2chr[i][0];
    blk2chr[i][1] = s->blk2chr[i][1];
    for (j=0;j<9;j++){
      map2chr[i][j] = s->map2chr[i][j];
      colpre[i][j] = s->colpre[i][j];
      rowpre[i][j] = s->rowpre[i][j];
    }
  }

  CUDA_ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_blk2chr), blk2chr, 9*2*sizeof(int)))

  CUDA_ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_map2chr), map2chr, 9*9*sizeof(int)))

  CUDA_ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_colpre), colpre, 9*9*sizeof(int)))

  CUDA_ERROR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_rowpre), rowpre, 9*9*sizeof(int)))

  DEBUG("\npannel_position(mapping)chromosome_index--constant\npresent bits by column & row--constant\n");

  // global mem  
  char * pp, * final;

  CUDA_ERROR_CHECK(hipMalloc((void **)&pp, N*s->chr_size*sizeof(char)))
  CUDA_ERROR_CHECK(hipMemcpy((void *)pp, pool, N*s->chr_size*sizeof(char), hipMemcpyHostToDevice))
  //printf("allocated population %d*%d*%d = %d pool--global\n", N, s->chr_size, sizeof(char));

  CUDA_ERROR_CHECK(hipMalloc((void **)&final, s->chr_size*sizeof(char)))
  DEBUG("allocated final char area--global\n");
  
  // init with random numbers, hiprand library
  hiprandState * allStates;
  hipMalloc(&allStates, N*sizeof(hiprandState));
  ini_device<<<BLKN, N>>>(allStates, time(NULL));

  DEBUG("initialized--hiprand\nsolving....");

  solve<<<BLKN, N>>>(allStates, pp, final, s->chr_size);

  char * result = (char *)malloc(sizeof(char)*s->chr_size);
  CUDA_ERROR_CHECK(hipMemcpy(result, final, s->chr_size*sizeof(char), hipMemcpyDeviceToHost))
  //for (i=0;i<s->chr_size;i++){printf("%c", result[i]);}
  //printf("\n");
  DEBUG("\nsolved--GPU\n");

  export_sudoku(result, s, sf);
  
  // Free
  free(result);
  hipFree(pp);
  hipFree(final);
  hipFree(allStates);
  
  free_exit(f, sf, pool, s);
  return 0;
}

void free_exit(FILE *f, FILE *sf, char * pool, sudoku_puzzle *s){
  fclose(f);
  fclose(sf);
  free(pool);
  clean_sudoku(s);
  exit(0);
}


void shuffle(int * array, int size){
  if (size>1){
    int i=0;
    for (; i<size-1; i++){
      int j = i + rand()/(RAND_MAX/(size-i) + 1);
      int t = array[j];
      array[j] = array[i];
      array[i] = t;
    }
  }
}

char * scan_puzzle(sudoku_puzzle *s, int pool_size){
  // scan and generate arrays
  int i, j;
  int ** map = (int **)malloc(sizeof(int *)*9);
  int ** map_chr = (int **)malloc(sizeof(int *)*9);
  int chr_size = 0;

  for (i=0; i<9; i++){
    // for each block

    int all[] = {1, 2, 3, 4, 5, 6, 7, 8, 9};
    int n = 0; // missing in this blocks

    for (j=0; j<9; j++){
      if (s->blocks[i][j].on){ // present
        all[s->blocks[i][j].val-1] = 0; // set present to 0
      }else{ // missing
        int r = s->blocks[i][j].i;
        int c = s->blocks[i][j].j;
        s->map2chr[r][c] = chr_size + n;
        n++;
      }
    }

    map[i] = (int *)malloc(sizeof(int)*n); // missing
    map_chr[i] = (int *)malloc(sizeof(int)*2); //start, length
    map_chr[i][0] = i==0?0:map_chr[i-1][0] + map_chr[i-1][1];
    map_chr[i][1] = n; 
    chr_size += n;

    int k = 0;
    for (j=0; j<9; j++){
      if (all[j] != 0){ // non-0->missing
        map[i][k] = all[j];
        k++;
      }
    }
  }
                                                               
  // randomize initial chromosomes pool

  srand(time(NULL));
  int * pool = (int *)malloc(sizeof(int)*pool_size*chr_size);
  for (i=0; i<N; i++){ 
    for (j=0; j<9; j++){
      // printf("block %d size %d chrstart %d\n", j, map_chr[j][1], map_chr[j][0]);
      shuffle(map[j], map_chr[j][1]);
      memcpy(pool+i*chr_size+map_chr[j][0], map[j], map_chr[j][1]*sizeof(int));
    }
  }
  char * pc = (char *)malloc(sizeof(char)*pool_size*chr_size+1);
  char tmp[2];
  for (i=0;i<pool_size*chr_size;i++){
    sprintf(tmp, "%d", pool[i]);
    memcpy(&pc[i], tmp, sizeof(char));
  }

  s->map = map;
  s->blk2chr = map_chr;
  s->chr_size = chr_size;
  
  // debug
  /*
  for(i=0;i<pool_size;i++){
    printf("chromosome %d: ",i);
    for(j=0;j<chr_size;j++){
      printf("%d", pc[i*chr_size+j]-'0');
      //printf("%d",pool[i*chr_size+j]);
    }
    printf("\n");
  }
  */
  // debug
  free(pool);
  return pc;
}

